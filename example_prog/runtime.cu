#include <iostream>
#include <hip/hip_runtime.h>

__global__ void foo_device(int * n){
	//int i = threadIdx.x;	
	n[0] = 42;
}

int main(int argc, char const *argv[])
{
	int * device;
	int host;
    hipError_t error;
    
	error = hipMalloc( (void **) &device, sizeof(int));
    
    if (error != hipSuccess)
    {
        std::cout << "hipMalloc returned error " << hipGetErrorString(error) << "\n";
    }
    
	foo_device<<<1,20>>>(device);
    
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cout << "kernel returned error " << hipGetErrorString(error) << "\n";;
    }
	error = hipMemcpy(&host, device, sizeof(int), hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        std::cout << "cudaMemcyp returned error " << hipGetErrorString(error) << "\n";;
    }
    
	std::cout << "the cuda number is: " << host << std::endl;
	return 0;
}
