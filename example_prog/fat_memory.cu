#include <iostream>
#include <hip/hip_runtime.h>
#include <unistd.h>

int main(int argc, char const *argv[])
{
	int * device;
	hipError_t error;
    
	error = hipMalloc( (void **) &device, sizeof(double)*4096*4096);
    
    	if (error != hipSuccess)
    	{
        	std::cout << "hipMalloc returned error " << hipGetErrorString(error) << "\n";
    	}
    
	sleep(3);

	std::cout << "time is over" << std::endl;
	
	return 0;
}
