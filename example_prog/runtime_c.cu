#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void foo_device(int * n){
	int i = threadIdx.x;	
	n[i] = 7*i;
}

int main(int argc, char const *argv[])
{
    int * device;
    hipError_t error;
    
    int host[4];
    error = hipMalloc( (void **) &device, sizeof(int)*4);
    
    if (error != hipSuccess)
    {
        printf("hipMalloc returned error %s\n", hipGetErrorString(error));
    }
    
    foo_device<<<1,4>>>(device);
    
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("kernel returned error %s\n", hipGetErrorString(error));
    }

    error = hipMemcpy(host, device, sizeof(int)*4, hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        printf("cudaMemcyp returned error %s\n", hipGetErrorString(error));
    }
    
    int sum = host[0] + host[1] + host[2] + host[3];

    if(sum == 42){
	    printf("The program works fine! The right anwser is: %i\n", sum); 
    }else{
	    printf("The answer is wrong. 42 was expected, but it is: %i\n", sum);
    }

    return 0;
}
