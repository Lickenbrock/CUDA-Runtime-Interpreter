#include <iostream>
#include <hip/hip_runtime.h>
#include <kernel.cu>

int main(int argc, char const *argv[])
{
    int * device;
    int * host;
    hipError_t error;
    
    host = new int[4];
    error = hipMalloc( (void **) &device, sizeof(int)*4);
    
    if (error != hipSuccess)
    {
        std::cout << "hipMalloc returned error " << hipGetErrorString(error) << "\n";
    }
    
    bar_device<int><<<1,4>>>(device);
    
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cout << "kernel returned error " << hipGetErrorString(error) << "\n";;
    }

    error = hipMemcpy(host, device, sizeof(int)*4, hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        std::cout << "cudaMemcyp returned error " << hipGetErrorString(error) << "\n";;
    }
    
    int sum = host[0] + host[1] + host[2] + host[3];

    if(sum == 42){
	std::cout << "The program works fine! The right anwser is: " << sum << std::endl; 
    }else{
	std::cout << "The answer is wrong. 42 was expected, but it is: " << sum << std::endl;
    }

    return 0;
}
